#include "mandelbrot.cuh"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    double x_min = -2.0, x_max = 1.0;
    double y_min = -1.5, y_max = 1.5;
    bool needsUpdate = true;
    sf::Vector2i lastMousePos;
    bool isDragging = false;

    unsigned char* d_output;
    unsigned char* h_output = new unsigned char[WIDTH * HEIGHT * 4];
    if (!h_output) {
        std::cerr << "Failed to allocate h_output" << std::endl;
        return 1;
    }
    hipError_t err = hipMalloc(&d_output, WIDTH * HEIGHT * 4 * sizeof(unsigned char));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed: " << hipGetErrorString(err) << std::endl;
        delete[] h_output;
        return 1;
    }

    sf::RenderWindow window;
    sf::Texture texture;
    sf::Sprite sprite;
    initWindow(window, texture, sprite);

    // Initial render
    computeMandelbrot(d_output, WIDTH, HEIGHT, x_min, x_max, y_min, y_max);
    hipMemcpy(h_output, d_output, WIDTH * HEIGHT * 4, hipMemcpyDeviceToHost);
    updateTexture(texture, h_output);

    while (window.isOpen()) {
        handleEvents(window, x_min, x_max, y_min, y_max, needsUpdate, lastMousePos, isDragging);

        if (needsUpdate) {
            computeMandelbrot(d_output, WIDTH, HEIGHT, x_min, x_max, y_min, y_max);
            err = hipMemcpy(h_output, d_output, WIDTH * HEIGHT * 4, hipMemcpyDeviceToHost);
            if (err != hipSuccess) {
                std::cerr << "CUDA memcpy failed: " << hipGetErrorString(err) << std::endl;
                break;
            }
            updateTexture(texture, h_output);
            needsUpdate = false;
        }

        window.clear();
        window.draw(sprite);
        window.display();
    }

    hipFree(d_output);
    delete[] h_output;
    return 0;
}